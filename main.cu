#ifdef _WIN32
#  define NOMINMAX 
#endif

#include "kernels.cu"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <fstream>
//#include <math>
#define NUMY 10
using std::ifstream;
using std::string;
using std::ofstream;

int hibit(unsigned int n) {
    n |= (n >>  1);
    n |= (n >>  2);
    n |= (n >>  4);
    n |= (n >>  8);
    n |= (n >> 16);
    return n - (n >> 1);
}

float* generateDeviceArray(int size){
    float* deviceArr;
    hipMalloc(deviceArr,size*sizeof(float));
    return deviceArr;
}

float* generateRandomWeights(int size){
    float* weightArr = (float*) malloc(size*sizeof(float));
    for(int i = 0; i < size;i++){
        weightArr[i] = .1;
    }
    return weightArr;
}


unsigned char* read_arrLabels(char* filename, int &len) {
    
    ifstream infile(filename);
    string line;
    int temp;
    getline(infile,line);
    temp = stoi(line);
    if(temp<len) len=temp;
    unsigned char *x = (unsigned char*) malloc(len * sizeof(char));
    for (int i = 0; i < len; i++) {
        //fscanf(fp, "%f", &x[i]);
        getline(infile,line);
        x[i] = stoi(line);
        printf("i = %d,x[i] = %d",i,x[i]);
    }
    infile.close();
    return x;
}
unsigned char* read_arrImage(char* filename, int &len,int &rows,int &cols) {
    //FILE *fp = fopen(filename, "r");
    ifstream infile(filename);
    int temp;
    string line;
    getline(infile,line);
    temp = stoi(line);
    //fscanf(fp,"%d",&temp);
    if(temp<len) len=temp;
    getline(infile,line);
    rows = stoi(line);
    getline(infile,line);
    cols = stoi(line);
    //fscanf(fp,"%d",&rows);
    //fscanf(fp,"%d",&cols);
    unsigned char *x = (unsigned char*) malloc(len*(rows)*(cols) * sizeof(char));
    for (int i = 0; i < len*(rows)*(cols); i++) {
        //fscanf(fp, "%f", &x[i]);
        getline(infile,line);
        x[i] = stoi(line);
    }
    infile.close();
    return x;
}

unsigned char* numToArr(char num){
    unsigned char* x = (unsigned char*) malloc(10*sizeof(unsigned char));
    for(int i = 0; i < 10;i++){
        if(i==num)x[i]=1;
        else x[i]=0;
    }
}


void trainingInstance(float* dx,float* dh, float* dy,float* dyCorrect,float* ddels,float* dgammas,float* dinter,float* dWeights1,float* dWeights2,float* ddeltas1,float* ddeltas2,int numX,int numH,int numY,float offset,float alpha,float lrate,int dinterSize){

    //firstLayer
    forwardPropagation<<<numH,numX>>>(dx,dinter,dWeights1,dinterSize,offset);
    printf("First forward propagation done\n");
    matrixReduction<<<numH,numX,numX*sizeof(float)>>>(dinter,dh,1024,hibit(1024));
    printf("First reduction done\n");
    sigmoidKernel<<<1,numH>>>(dh);
    printf("First sigmoid done\n");
    //first layer done

    //second layer:
    forwardPropagation<<<numY,numH>>>(dh,dinter,dWeights2,dinterSize,offset);
    printf("second forward propagation done\n");
    matrixReduction<<<numY,numH,numH*sizeof(float)>>>(dinter,dy,1024,hibit(1024));
    printf("second reduction done\n");

    sigmoidKernel<<<1,numY>>>(dy);
    printf("second sigmoid done\n");
    //second layer done

    //backpropagation:
    

    backPropagationFirstKernel<<<numY,numH>>>(dh,dy,dyCorrect,dWeights2,ddeltas2,ddels,alpha,lrate);
    //dim3 grid(numY,numH);
    backPropagationSecondKernelPart1<<<numY,numH>>>(dh,dgammas,dWeights1,ddels,alpha,lrate);
    matrixReduction<<<numH,numY,numY*sizeof(float)>>>(dgammas,dgammas,numY,hibit(numY));
    backPropagationSecondKernelPart2<<<numH,numX>>>(dx,dgammas,dWeights1,ddeltas1,alpha,lrate);
}

int main(int argc,char** argv){


    int debugLine = 0;
    unsigned char* trainImage;
    unsigned char* trainLabels;
    int len = 1;
    int rows;
    int cols;





    
    //printf("Got to debug # %d\n",++debugLine);
    
    trainImage = read_arrImage("imagesTrain.txt",len,rows,cols);
    printf("Len: %d\nRows: %d\nCols: %d\n",len,rows,cols);
    // for(int i = 0; i < rows;i++){
    //     for(int j = 0; j < cols;j++){
    //         printf("%d ",trainImage[i*cols+j]);
    //     }
    //     printf("\n");
    // }
    len = 1;



    trainLabels = read_arrLabels("labelsTrain.txt",len);
    printf("Len: %d\n",len);
    // for(int i = 0; i < 10;i++){
    //     printf("trainLabels[%d]: %d\n",i,trainLabels[i]);
    // }


    int numX = rows*cols;
    int numY = NUMY;
    int numH = 500;
    float* dx = generateDeviceArray(rows*cols);
    hipMemcpy(dx,trainImage,rows*cols*sizeof(float),hipMemcpyHostToDevice);
    float* dh = generateDeviceArray(numH);
    float* dy = generateDeviceArray(NUMY);
    float* dyCorrect = generateDeviceArray(NUMY);
    float* hyCorrect = numToArr(trainLabels[0]);
    hipMemcpy(dyCorrect,hyCorrect,NUMY*sizeof(float),hipMemcpyHostToDevice);
    float* ddels = generateDeviceArray(NUMY);
    float* dgammas = generateDeviceArray(numH*NUMY);
    float* dinter = generateDeviceArray(1024*1024);
    float* hWeights1 = generateRandomWeights(numX*numH)
    float* dWeights1 = generateDeviceArray(numX*numH);
    hipMemcpy(dWeights1,hWeights1,numX*numH*sizeof(float),hipMemcpyHostToDevice);
    float* hWeights2 = generateRandomWeights(numH*NUMY);
    float* dWeights2 = generateDeviceArray(numH*NUMY);
    hipMemcpy(dWeights2,hWeights2,numH*NUMY*sizeof(float),hipMemcpyHostToDevice);
    float* ddeltas1 = generateDeviceArray(rows*cols*numH);
    float* ddeltas2 = generateDeviceArray(numH*NUMY);
    float alpha = .1;
    float lrate = .1;
    int dinterSize = 1024;
    float offset = 1;

    trainingInstance(dx,dh,dy,dyCorrect,ddels,dgammas,dinter,dWeights1,dWeights2,ddeltas1,ddeltas2,numX,numH,numY,offset,alpha,lrate,dinterSize);






    free(trainLabels);
    free(trainImage);
    
    //Initialize weight matrices

    //get inputs from training file
    //get inputs from test file

    //todo: add main
    /*float* a = (float*) malloc(2*13*sizeof(float));
    float* b = (float*) malloc(2);
    for(int i = 0; i < 13;i++){
        a[i] = i;
        b[0] +=i;
        a[i+13] = i;
        b[1]+=i;
    }
    a[13] +=100;
    b[1]+=100;
    float* da;
    printf("hibit: %x\n",hibit(13));
    hipMalloc(&da,sizeof(float)*26);
    hipMemcpy(da,a,sizeof(float)*26,hipMemcpyHostToDevice);
    matrixReductionDestructive<<<2,13,13*sizeof(float)>>>(da,13,hibit(13)<<1);
    hipMemcpy(a,da,sizeof(float)*26,hipMemcpyDeviceToHost);

    printf("Device Results: %f,%f\n",a[0],a[13]);
    printf("Host Results: %f,%f\n",b[0],b[1]);
    hipFree(da);
    free(a);
    free(b);
    */
}



