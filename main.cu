#ifdef _WIN32
#  define NOMINMAX 
#endif

#include "kernels.cu"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <string>
#include <fstream>
#include <random>
//#include <math>
#define NUMY 10
#include <time.h>
using std::ifstream;
using std::string;
using std::ofstream;

int hibit(unsigned int n) {
    n |= (n >>  1);
    n |= (n >>  2);
    n |= (n >>  4);
    n |= (n >>  8);
    n |= (n >> 16);
    return n - (n >> 1);
}

double* generateDeviceArray(int size){
    double* deviceArr;
    hipMalloc(&deviceArr,size*sizeof(double));
    return deviceArr;
}

double* generateRandomWeights(int size){
    double* weightArr = (double*) malloc(size*sizeof(double));
    std::default_random_engine generator;
    std::uniform_real_distribution<double> distribution(-.1,.1);
    for(int i = 0; i < size;i++){
        weightArr[i] = distribution(generator);
    }
    return weightArr;
}

void printArr(double* arr,int rows,int cols,std::string s){
    std::cout<<s<<std::endl;
    //printf("\n");
    for(int i = 0; i < rows;i++){
        for(int j = 0; j < cols;j++){
            printf(" %lf ", arr[i*cols+j] );
        }
        printf("\n");
    }
}

void printArrFromDevice(double* darr,int rows,int cols,std::string s){
    double* harr = (double*) malloc(rows*cols*sizeof(double));
    hipMemcpy(harr,darr,rows*cols*sizeof(double),hipMemcpyDeviceToHost);
    printArr(harr,rows,cols,s);
    free(harr);
}

double* read_arrLabels(char* filename, int &len) {
    
    ifstream infile(filename);
    string line;
    int temp;
    getline(infile,line);
    temp = stoi(line);
    if(temp<len) len=temp;
    int tempint = 0;
    double *x = (double*) malloc(10*len * sizeof(double));
    for (int i = 0; i < len; i++) {
        //fscanf(fp, "%f", &x[i]);
        getline(infile,line);
        tempint = stoi(line);
        //printf("tempint: %d\n",tempint);
        for(int j = 0; j < 10;j++){
            if(tempint == j) {
                x[i*10+j] = 1.0;
            }
            else
            {
                x[i*10+j]=0.0;
            }
            //printf("x[%d*10+%d] = %lf",i,j,x[i*10+j]);
        }
        //printf("i = %d,x[i] = %d",i,x[i]);
    }
    infile.close();
    return x;
}

double* read_arrLabelsTest(char* filename, int &len,int* correct) {
    
    ifstream infile(filename);
    string line;
    int temp;
    getline(infile,line);
    temp = stoi(line);
    if(temp<len) len=temp;
    int tempint = 0;
    double *x = (double*) malloc(10*len * sizeof(double));
    correct = (int*)malloc(len*sizeof(int));
    for (int i = 0; i < len; i++) {
        //fscanf(fp, "%f", &x[i]);
        getline(infile,line);
        tempint = stoi(line);
        correct[i] = tempint;
        //printf("tempint: %d\n",correct[i]);
        for(int j = 0; j < 10;j++){
            if(tempint == j) {
                x[i*10+j] = 1.0;
            }
            else
            {
                x[i*10+j]=0.0;
            }
            //printf("x[%d*10+%d] = %lf",i,j,x[i*10+j]);
        }
        //printf("i = %d,x[i] = %d",i,x[i]);
    }
    infile.close();
    return x;
}
char* read_arrImage(char* filename, int &len,int &rows,int &cols) {
    //FILE *fp = fopen(filename, "r");
    ifstream infile(filename);
    int temp;
    string line;
    getline(infile,line);
    temp = stoi(line);
    //fscanf(fp,"%d",&temp);
    if(temp<len) len=temp;
    getline(infile,line);
    rows = stoi(line);
    getline(infile,line);
    cols = stoi(line);
    //fscanf(fp,"%d",&rows);
    //fscanf(fp,"%d",&cols);
    char *x = (char*) malloc(len*(rows)*(cols) * sizeof(char));
    for (int i = 0; i < len*(rows)*(cols); i++) {
        //fscanf(fp, "%f", &x[i]);
        getline(infile,line);
        x[i] = stoi(line);
    }
    infile.close();
    return x;
}

/*
double* numToArr(char num){
    double* x = (double*) malloc(10*sizeof(double));
    for(int i = 0; i < 10;i++){
        if(i==num)x[i]=1;
        else x[i]=0;
    }

}
*/

void trainingInstance(double* dx,double* dh, double* dy,double* dyCorrect,double* ddels,double* dgammas,double* dinter,double* dWeights1,double* dWeights2,double* ddeltas1,double* ddeltas2,int numX,int numH,int numY,double offset,double alpha,double lrate,int dinterSize){
    //double* testOutput = (double*)malloc(10*sizeof(double));
    //firstLayer
    // printArrFromDevice(dx,1,numX);
    forwardPropagation<<<numH,numX>>>(dx,dinter,dWeights1,dinterSize,0);
    // printf("dinter\n");
    // printArrFromDevice(dinter,2,1024);
    // printArrFromDevice(dWeights1,numH,numX);
    // printf("First forward propagation done\n");
    matrixReductionToVector<<<numH,numX,numX*sizeof(double)>>>(dinter,dh,1024,hibit(1024));
    // printArrFromDevice(dh,1,numH);
    // printf("First reduction done\n");

    sigmoidKernel<<<1,numH>>>(dh);
    // printf("First sigmoid done\n");
    //first layer done
    // printArrFromDevice(dh,1,numH);
    //second layer:
    forwardPropagation<<<numY,numH>>>(dh,dinter,dWeights2,dinterSize,offset);
    // printf("second forward propagation done\n");
    // printArrFromDevice(dWeights2,numY,numH);
    matrixReductionToVector<<<numY,numH,numH*sizeof(double)>>>(dinter,dy,1024,hibit(1024));
    // printf("second reduction done\n");
    // printArrFromDevice(dy,1,numY);
    sigmoidKernel<<<1,numY>>>(dy);
    // printArrFromDevice(dy,1,numY);
    // printf("second sigmoid done\n");
    //second layer done

    //backpropagation:
    
    // printf("dyCorrect then dy\n");
    //printArrFromDevice(dy,1,numY,"OutputY: ");
    //printArrFromDevice(dyCorrect,1,numY,"CorrectY ");
    //printArrFromDevice(dy,1,numY);
    backPropagationFirstKernel<<<numY,numH>>>(dh,dy,dyCorrect,dWeights2,ddeltas2,ddels,alpha,lrate);
    //printf("Deltas for W2: \n");
    //printArrFromDevice(ddeltas2,numY,numH);
    //dim3 grid(numY,numH);
    //printf("Dels: \n");
    //printArrFromDevice(ddels,1,numY);
    backPropagationSecondKernelPart1<<<numY,numH>>>(dh,dgammas,dWeights1,ddels,alpha,lrate);
    //printf("dgammas\n");

    //printArrFromDevice(dgammas,numH,numY);

    matrixReduction<<<numH,numY,numY*sizeof(double)>>>(dgammas,dgammas,numY,hibit(numY));
    backPropagationSecondKernelPart2<<<numH,numX>>>(dx,dgammas,dWeights1,ddeltas1,alpha,lrate);
    //printf("Deltas for W1: \n");
    //printArrFromDevice(ddeltas1,numH,numX,"W1 Deltas");
    //free(testOutput);
}


void longTraining(int len,double* trainLabels,char* trainImage,int epochs,double* dx,double* dh, double* dy,double* dyCorrect,double* ddels,double* dgammas,double* dinter,double* dWeights1,double* dWeights2,double* ddeltas1,double* ddeltas2,int numX,int numH,int numY,double offset,double alpha,double lrate,int dinterSize){


    double* trainImageDouble = (double*) malloc(numX*sizeof(double));
    double* trainLabelsInner = (double*) malloc(numY*sizeof(double));
    for(int q = 0; q < epochs;q++){
        for(int i = 0; i < len;i++){
            for(int j = 0; j < numX;j++){
                trainImageDouble[j] = (double)trainImage[j+i*numX];
            }
            for(int j = 0; j < numY;j++){
                trainLabelsInner[j] = trainLabels[j+i*numY];
                //printf(" %lf ",trainLabelsInner[j]);

            }
            //printf("\n");
            hipMemcpy(dx,trainImageDouble,numX*sizeof(double),hipMemcpyHostToDevice);
            //free(hyCorrect);
            //hyCorrect = numToArr(trainLabels[i]);
            hipMemcpy(dyCorrect,trainLabelsInner,numY*sizeof(double),hipMemcpyHostToDevice);
            trainingInstance(dx,dh,dy,dyCorrect,ddels,dgammas,dinter,dWeights1,dWeights2,ddeltas1,ddeltas2,numX,numH,numY,offset,alpha,lrate,dinterSize);
        }
    }
    //free(hyCorrect);
    free(trainImageDouble);
    free(trainLabelsInner);
}

void testingInstance(double* dx,double* dh, double* dy,double* dinter,double* dWeights1,double* dWeights2,int numX,int numH,int numY,double offset,int dinterSize){
    forwardPropagation<<<numH,numX>>>(dx,dinter,dWeights1,dinterSize,0);
    matrixReductionToVector<<<numH,numX,numX*sizeof(double)>>>(dinter,dh,1024,hibit(1024));
    sigmoidKernel<<<1,numH>>>(dh);
    forwardPropagation<<<numY,numH>>>(dh,dinter,dWeights2,dinterSize,offset);
    matrixReductionToVector<<<numY,numH,numH*sizeof(double)>>>(dinter,dy,1024,hibit(1024));
    sigmoidKernel<<<1,numY>>>(dy);
}

void testing(int len,double* testLabels,char* testImage,double* results,double* dx,double* dh, double* dy,double* dinter,double* dWeights1,double* dWeights2,int numX,int numH,int numY,double offset,int dinterSize){

    double* testImageDouble = (double*) malloc(numX*sizeof(double));
    double* testLabelsInner = (double*) malloc(numY*sizeof(double));
    for(int i = 0; i < len;i++){
        for(int j = 0; j < numX;j++){
                testImageDouble[j] = (double)testImage[j+i*numX];
            }
        for(int j = 0; j < numY;j++){
                testLabelsInner[j] = testLabels[j+i*numY];
                //printf(" %lf ",trainLabelsInner[j]);
            }
            //printf("\n");
        hipMemcpy(dx,testImageDouble,numX*sizeof(double),hipMemcpyHostToDevice);
            //free(hyCorrect);
            //hyCorrect = numToArr(trainLabels[i]);
        //hipMemcpy(dyCorrect,testLabelsInner,numY*sizeof(double),hipMemcpyHostToDevice);
        testingInstance(dx,dh,dy,dinter,dWeights1,dWeights2,numX,numH,numY,offset,dinterSize);

        hipMemcpy(testLabelsInner,dy,numY*sizeof(double),hipMemcpyDeviceToHost);
        for(int j = 0;j < numY;j++){
            results[j+i*numY] = testLabelsInner[j];
        }
    }
    free(testImageDouble);
    free(testLabelsInner);
}




int main(int argc,char** argv){


    char* trainImage;
    double* trainLabels;
    int len = 10000;
    int rows;
    int cols;
	srand(time(NULL));
    char* testImage;
    double* testLabels;
    int* correct;
    int epochs = std::stoi(argv[1]);

    
    //printf("Got to debug # %d\n",++debugLine);
    //trainImage = (char* )malloc(10*sizeof(char));
    trainImage = read_arrImage("imagesTrain.txt",len,rows,cols);
    printf("Len: %d\nRows: %d\nCols: %d\n",len,rows,cols);
    /*
    for(int i = 0; i < rows;i++){
        for(int j = 0; j < cols;j++){
            printf("%d ",trainImage[i*cols+j]);
        }
        printf("\n");
    }
    */
    len = 10000;



    trainLabels = read_arrLabels("labelsTrain.txt",len);
    //trainLabels = (double*) malloc(2*sizeof(double));
    printf("Len: %d\n",len);


    /*
    for(int i = 0; i < len;i++){
        printf("trainLabels[%d]: \n",i);
        for(int j = 0; j < NUMY;j++){
            printf(" %lf ",trainLabels[i*NUMY+j]);
        }
        printf("\n");
        
    }
    */
    int testLen = 10000;
    testImage = read_arrImage("imagesTest.txt",testLen,rows,cols);
    testLabels = read_arrLabels("labelsTest.txt",testLen);
    correct = (int*) malloc(testLen*sizeof(int));

	for(int i = 0; i < testLen;i++){
		for(int j =0 ; j < NUMY;j++){
		if(testLabels[j+i*NUMY]>.9){
		correct[i] = j;
			break;
			}	
		}
	}
    //int numX = 10;
    printf("TestLen: %d\n",testLen);
	printf("Testing For correct: %d\n",correct[0]);

    int numX = rows*cols;
    int numY = NUMY;
    int numH = 50;


    
    double* dx = generateDeviceArray(numX);
    hipMemcpy(dx,trainImage,numX*sizeof(double),hipMemcpyHostToDevice);
    //hipMemcpy(dx,trainImageDouble,numX*sizeof(double),hipMemcpyHostToDevice);
    //free(trainImageDouble);
    double* dh = generateDeviceArray(numH);
    double* dy = generateDeviceArray(NUMY);
    double* dyCorrect = generateDeviceArray(NUMY);
    double* hyCorrect = (double*)malloc(numY*sizeof(double));//numToArr(trainLabels[0]);
    //hipMemcpy(dyCorrect,hyCorrect,NUMY*sizeof(double),hipMemcpyHostToDevice);
    double* ddels = generateDeviceArray(NUMY);
    double* dgammas = generateDeviceArray(numH*NUMY);
    double* dinter = generateDeviceArray(1024*1024);
    double* hWeights1 = generateRandomWeights(numX*numH);
    //printArr(hWeights1,numH,numX,"");
    double* dWeights1 = generateDeviceArray(numX*numH);
    hipMemcpy(dWeights1,hWeights1,numX*numH*sizeof(double),hipMemcpyHostToDevice);
    double* hWeights2 = generateRandomWeights(numH*NUMY);
    double* dWeights2 = generateDeviceArray(numH*NUMY);
    hipMemcpy(dWeights2,hWeights2,numH*NUMY*sizeof(double),hipMemcpyHostToDevice);
    double* ddeltas1 = generateDeviceArray(numX*numH);
    double* ddeltas2 = generateDeviceArray(numH*NUMY);
    double alpha = .1;
    double lrate = .01;
    int dinterSize = 1024;
    double offset = .1;

    double* results =(double*) malloc(testLen*NUMY*sizeof(double));
    int* bestMatch = (int*)malloc(testLen*sizeof(int));


    longTraining(len,trainLabels,trainImage,epochs,dx,dh,dy,dyCorrect,ddels,dgammas,dinter,dWeights1,dWeights2,ddeltas1,ddeltas2,numX,numH,numY,offset,alpha,lrate,dinterSize);

    testing(testLen,testLabels,testImage,results,dx,dh,dy,dinter,dWeights1,dWeights2,numX,numH,numY,offset,dinterSize);
	
    int* dbestMatch;
	hipMalloc(&dbestMatch,testLen*sizeof(int));
    //double* dresults= 
	double* dresults =     generateDeviceArray(testLen*NUMY);

    int numThreads = 1024;
    int numBlocks = testLen/1024 + 1;
hipMemcpy(dresults,results,sizeof(double)*testLen*NUMY,hipMemcpyHostToDevice);
    bestChoiceKernel<<<numBlocks,numThreads>>>(dresults,dbestMatch,testLen,NUMY);
   hipMemcpy(bestMatch,dbestMatch,sizeof(int)*testLen,hipMemcpyDeviceToHost);
    int err = 0;
    int right = 0;
  // int temp = bestMatch[0];
//double temp2 = results[0];
//int temp = correct[0];
//printf("%d\n",temp);
//printf("%lf\n",temp2);
//printf("%d\n",correct[0]);

for(int i =0; i < 10;i++){
    for(int j = 0; j < NUMY;j++){
	   printf(" %f ",results[j+NUMY*i]);
    }
    printf("\n");
}

    for(int i = 0; i < testLen;i++){
    if(i<10){
       printf("best: %d\ncorrect: %d\n",bestMatch[i],correct[i]);
	
    }
        if(bestMatch[i]!=correct[i]){
            err++;
        }
        else{
            right++;
        }
    }
  
    printf("# correct: %d\n",right);
    printf("# wrong: %d\n",err);
    //trainingInstance(dx,dh,dy,dyCorrect,ddels,dgammas,dinter,dWeights1,dWeights2,ddeltas1,ddeltas2,numX,numH,numY,offset,alpha,lrate,dinterSize);




    free(hyCorrect);
    free(hWeights2);
    free(hWeights1);
    hipFree(dx);
    hipFree(dh);
    hipFree(dy);
    hipFree(dyCorrect);
    hipFree(ddels);
    hipFree(dgammas);
    hipFree(dinter);
    hipFree(ddeltas1);
    hipFree(ddeltas2);
    hipFree(dWeights2);
    hipFree(dWeights1);
hipFree(dresults);
hipFree(dbestMatch);
    free(testLabels);
    free(testImage);
	free(correct);
free(results);
free(bestMatch);
    free(trainLabels);
    free(trainImage);
    
    //Initialize weight matrices

    //get inputs from training file
    //get inputs from test file

    //todo: add main
    /*double* a = (double*) malloc(2*13*sizeof(double));
    double* b = (double*) malloc(2);
    for(int i = 0; i < 13;i++){
        a[i] = i;
        b[0] +=i;
        a[i+13] = i;
        b[1]+=i;
    }
    a[13] +=100;
    b[1]+=100;
    double* da;
    printf("hibit: %x\n",hibit(13));
    hipMalloc(&da,sizeof(double)*26);
    hipMemcpy(da,a,sizeof(double)*26,hipMemcpyHostToDevice);
    matrixReductionDestructive<<<2,13,13*sizeof(double)>>>(da,13,hibit(13)<<1);
    hipMemcpy(a,da,sizeof(double)*26,hipMemcpyDeviceToHost);

    printf("Device Results: %f,%f\n",a[0],a[13]);
    printf("Host Results: %f,%f\n",b[0],b[1]);
    hipFree(da);
    free(a);
    free(b);
    */
}



