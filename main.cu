#ifdef _WIN32
#  define NOMINMAX 
#endif

#include "kernels.cu"
#include <stdio.h>
#include <stdlib.h>
#include <string>
#include <fstream>
#include <random>
//#include <math>
#define NUMY 2
using std::ifstream;
using std::string;
using std::ofstream;

int hibit(unsigned int n) {
    n |= (n >>  1);
    n |= (n >>  2);
    n |= (n >>  4);
    n |= (n >>  8);
    n |= (n >> 16);
    return n - (n >> 1);
}

double* generateDeviceArray(int size){
    double* deviceArr;
    hipMalloc(&deviceArr,size*sizeof(double));
    return deviceArr;
}

double* generateRandomWeights(int size){
    double* weightArr = (double*) malloc(size*sizeof(double));
    std::default_random_engine generator;
    std::uniform_real_distribution<double> distribution(-.1,.1);
    for(int i = 0; i < size;i++){
        weightArr[i] = distribution(generator);
    }
    return weightArr;
}

void printArr(double* arr,int rows,int cols){
    printf("PRINTING NEW ARRAY!\n");
    for(int i = 0; i < rows;i++){
        for(int j = 0; j < cols;j++){
            printf(" %lf ", arr[i*cols+j] );
        }
        printf("\n");
    }
}

void printArrFromDevice(double* darr,int rows,int cols){
    double* harr = (double*) malloc(rows*cols*sizeof(double));
    hipMemcpy(harr,darr,rows*cols*sizeof(double),hipMemcpyDeviceToHost);
    printArr(harr,rows,cols);
    free(harr);
}

unsigned char* read_arrLabels(char* filename, int &len) {
    
    ifstream infile(filename);
    string line;
    int temp;
    getline(infile,line);
    temp = stoi(line);
    if(temp<len) len=temp;
    unsigned char *x = (unsigned char*) malloc(len * sizeof(char));
    for (int i = 0; i < len; i++) {
        //fscanf(fp, "%f", &x[i]);
        getline(infile,line);
        x[i] = stoi(line);
        //printf("i = %d,x[i] = %d",i,x[i]);
    }
    infile.close();
    return x;
}
unsigned char* read_arrImage(char* filename, int &len,int &rows,int &cols) {
    //FILE *fp = fopen(filename, "r");
    ifstream infile(filename);
    int temp;
    string line;
    getline(infile,line);
    temp = stoi(line);
    //fscanf(fp,"%d",&temp);
    if(temp<len) len=temp;
    getline(infile,line);
    rows = stoi(line);
    getline(infile,line);
    cols = stoi(line);
    //fscanf(fp,"%d",&rows);
    //fscanf(fp,"%d",&cols);
    unsigned char *x = (unsigned char*) malloc(len*(rows)*(cols) * sizeof(char));
    for (int i = 0; i < len*(rows)*(cols); i++) {
        //fscanf(fp, "%f", &x[i]);
        getline(infile,line);
        x[i] = stoi(line);
    }
    infile.close();
    return x;
}

double* numToArr(char num){
    double* x = (double*) malloc(10*sizeof(double));
    for(int i = 0; i < 10;i++){
        if(i==num)x[i]=1;
        else x[i]=0;
    }
}


void trainingInstance(double* dx,double* dh, double* dy,double* dyCorrect,double* ddels,double* dgammas,double* dinter,double* dWeights1,double* dWeights2,double* ddeltas1,double* ddeltas2,int numX,int numH,int numY,double offset,double alpha,double lrate,int dinterSize){
    double* testOutput = (double*)malloc(10*sizeof(double));
    //firstLayer
    printArrFromDevice(dx,1,numX);
    forwardPropagation<<<numH,numX>>>(dx,dinter,dWeights1,dinterSize,0);
    printf("dinter\n");
    printArrFromDevice(dinter,2,1024);
    printArrFromDevice(dWeights1,numH,numX);
    printf("First forward propagation done\n");
    matrixReduction<<<numH,numX,numX*sizeof(double)>>>(dinter,dh,1024,hibit(1024));
    printArrFromDevice(dh,1,numH);
    printf("First reduction done\n");

    sigmoidKernel<<<1,numH>>>(dh);
    printf("First sigmoid done\n");
    //first layer done
    printArrFromDevice(dh,1,numH);
    //second layer:
    forwardPropagation<<<numY,numH>>>(dh,dinter,dWeights2,dinterSize,offset);
    printf("second forward propagation done\n");
    printArrFromDevice(dWeights2,numY,numH);
    matrixReduction<<<numY,numH,numH*sizeof(double)>>>(dinter,dy,1024,hibit(1024));
    printf("second reduction done\n");
    printArrFromDevice(dy,1,numY);
    sigmoidKernel<<<1,numY>>>(dy);
    printArrFromDevice(dy,1,numY);
    printf("second sigmoid done\n");
    //second layer done

    //backpropagation:
    

    backPropagationFirstKernel<<<numY,numH>>>(dh,dy,dyCorrect,dWeights2,ddeltas2,ddels,alpha,lrate);
    //dim3 grid(numY,numH);
    backPropagationSecondKernelPart1<<<numY,numH>>>(dh,dgammas,dWeights1,ddels,alpha,lrate);
    matrixReduction<<<numH,numY,numY*sizeof(double)>>>(dgammas,dgammas,numY,hibit(numY));
    backPropagationSecondKernelPart2<<<numH,numX>>>(dx,dgammas,dWeights1,ddeltas1,alpha,lrate);
    free(testOutput);
}

int main(int argc,char** argv){


    int debugLine = 0;
    unsigned char* trainImage;
    unsigned char* trainLabels;
    int len = 1;
    int rows;
    int cols;





    
    //printf("Got to debug # %d\n",++debugLine);
    trainImage = (unsigned char* )malloc(10*sizeof(unsigned char));
    //trainImage = read_arrImage("imagesTrain.txt",len,rows,cols);
    //printf("Len: %d\nRows: %d\nCols: %d\n",len,rows,cols);
    // for(int i = 0; i < rows;i++){
    //     for(int j = 0; j < cols;j++){
    //         printf("%d ",trainImage[i*cols+j]);
    //     }
    //     printf("\n");
    // }
    len = 1;



    trainLabels = read_arrLabels("labelsTrain.txt",len);
    //trainLabels = (double*) malloc(2*sizeof(double));
    printf("Len: %d\n",len);
    // for(int i = 0; i < 10;i++){
    //     printf("trainLabels[%d]: %d\n",i,trainLabels[i]);
    // }
    
    double* trainImageDouble = (double*)malloc(10*sizeof(double));
    for(int i = 0; i < 10;i++){
        trainImageDouble[i] = i;
    }
    int numX = 10;
    //int numX = rows*cols;
    int numY = NUMY;
    int numH = 2;
    double* dx = generateDeviceArray(numX);
    //hipMemcpy(dx,trainImage,rows*cols*sizeof(double),hipMemcpyHostToDevice);
    hipMemcpy(dx,trainImageDouble,numX*sizeof(double),hipMemcpyHostToDevice);
    free(trainImageDouble);
    double* dh = generateDeviceArray(numH);
    double* dy = generateDeviceArray(NUMY);
    double* dyCorrect = generateDeviceArray(NUMY);
    double* hyCorrect = (double*) malloc(2*sizeof(double));
    hyCorrect[0] = 0;
    hyCorrect[5] = 5;
    //double* hyCorrect = numToArr(trainLabels[0]);
    hipMemcpy(dyCorrect,hyCorrect,NUMY*sizeof(double),hipMemcpyHostToDevice);
    double* ddels = generateDeviceArray(NUMY);
    double* dgammas = generateDeviceArray(numH*NUMY);
    double* dinter = generateDeviceArray(1024*1024);
    double* hWeights1 = generateRandomWeights(numX*numH);
    printArr(hWeights1,numH,numX);
    double* dWeights1 = generateDeviceArray(numX*numH);
    hipMemcpy(dWeights1,hWeights1,numX*numH*sizeof(double),hipMemcpyHostToDevice);
    double* hWeights2 = generateRandomWeights(numH*NUMY);
    double* dWeights2 = generateDeviceArray(numH*NUMY);
    hipMemcpy(dWeights2,hWeights2,numH*NUMY*sizeof(double),hipMemcpyHostToDevice);
    double* ddeltas1 = generateDeviceArray(numX*numH);
    double* ddeltas2 = generateDeviceArray(numH*NUMY);
    double alpha = .1;
    double lrate = .1;
    int dinterSize = 1024;
    double offset = 1;

    trainingInstance(dx,dh,dy,dyCorrect,ddels,dgammas,dinter,dWeights1,dWeights2,ddeltas1,ddeltas2,numX,numH,numY,offset,alpha,lrate,dinterSize);




    free(hyCorrect);
    free(hWeights2);
    free(hWeights1);
    hipFree(dx);
    hipFree(dh);
    hipFree(dy);
    hipFree(dyCorrect);
    hipFree(ddels);
    hipFree(dgammas);
    hipFree(dinter);
    hipFree(ddeltas1);
    hipFree(ddeltas2);
    hipFree(dWeights2);
    hipFree(dWeights1);


    free(trainLabels);
    free(trainImage);
    
    //Initialize weight matrices

    //get inputs from training file
    //get inputs from test file

    //todo: add main
    /*double* a = (double*) malloc(2*13*sizeof(double));
    double* b = (double*) malloc(2);
    for(int i = 0; i < 13;i++){
        a[i] = i;
        b[0] +=i;
        a[i+13] = i;
        b[1]+=i;
    }
    a[13] +=100;
    b[1]+=100;
    double* da;
    printf("hibit: %x\n",hibit(13));
    hipMalloc(&da,sizeof(double)*26);
    hipMemcpy(da,a,sizeof(double)*26,hipMemcpyHostToDevice);
    matrixReductionDestructive<<<2,13,13*sizeof(double)>>>(da,13,hibit(13)<<1);
    hipMemcpy(a,da,sizeof(double)*26,hipMemcpyDeviceToHost);

    printf("Device Results: %f,%f\n",a[0],a[13]);
    printf("Host Results: %f,%f\n",b[0],b[1]);
    hipFree(da);
    free(a);
    free(b);
    */
}



