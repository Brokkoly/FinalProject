#ifdef _WIN32
#  define NOMINMAX 
#endif

#include "kernels.cu"
#include <stdio.h>
#include <stdlib.h>
int main(int argc,char** argv){

    //Initialize weight matrices

    //get inputs from training file
    //get inputs from test file

    //todo: add main
    float* a = (float*) malloc(2*13*sizeof(float));
    float* b = (float*) malloc(2);
    for(int i = 0; i < 13;i++){
        a[i] = i;
        b[0] +=i;
        a[i+13] = i;
        b[1]+=i;
    }
    a[13] +=100;
    b[1]+=100;
    float* da;
    hipMalloc((void**)&da,sizeof(float)*26);
    hipMemcpy(da,a,sizeof(float)*26,hipMemcpyHostToDevice);
    matrixReduction<<<2,13,13*sizeof(float)>>>(da,da);
    hipMemcpy(a,da,sizeof(float)*26,hipMemcpyDeviceToHost);

    printf("Device Results: %f,%f\nHost Results: %f,%f\n",a[0],a[13],b[0],b[1]);
    hipFree(da);
    free(a);
    free(b);



}

void trainingInstance(float* dx,float* dh, float* dy,float* dyCorrect,float* ddels,float* dgammas,float* dinter,float** dWeights,float** ddeltas,int numX,int numH,int numY,float alpha,float lrate,int dinterSize){

    //firstLayer
    forwardPropagation<<<numH,numX>>>(dx,dinter,dWeights[0],dinterSize);
    matrixReduction<<<numH,numX,numX*sizeof(float)>>>(dinter,dh);
    sigmoidKernel<<<1,numH>>>(dh);

    //first layer done

    //second layer:
    forwardPropagation<<<numY,numH>>>(dh,dinter,dWeights[1],dinterSize);
    matrixReduction<<<numY,numH,numH*sizeof(float)>>>(dinter,dy);
    sigmoidKernel<<<1,numY>>>(dy);

    //second layer done

    //backpropagation:
    

    backPropagationFirstKernel<<<numY,numH>>>(dh,dy,dyCorrect,dWeights[1],ddeltas[1],ddels,alpha,lrate);
    //dim3 grid(numY,numH);
    backPropagationSecondKernelPart1<<<numY,numH>>>(dh,dgammas,dWeights[1],ddels,alpha,lrate);
    matrixReduction<<<numH,numY,numY*sizeof(float)>>>(dgammas,dgammas);
    backPropagationSecondKernelPart2<<<numH,numX>>>(dx,dgammas,dWeights[0],ddeltas[0],alpha,lrate);






}
