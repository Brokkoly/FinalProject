#ifdef _WIN32
#  define NOMINMAX 
#endif

#include "kernels.cu"

int main(int argc,char** argv){

    //Initialize weight matrices

    //get inputs from training file
    //get inputs from test file

    //todo: add main





}

void trainingInstance(float* dx,float* dh, float* dy,float* dyCorrect,float* ddels,float* dgammas,float* dinter,float** dWeights,float** ddeltas,int numX,int numH,int numY,float alpha,float lrate,int dinterSize){

    //firstLayer
    forwardPropagation<<<numH,numX>>>(dx,dinter,dWeights[0],dinterSize);
    matrixReduction<<<numH,numX,numX*sizeof(float)>>>(dinter,dh);
    sigmoidKernel<<<1,numH>>>(dh);

    //first layer done

    //second layer:
    forwardPropagation<<<numY,numH>>>(dh,dinter,dWeights[1],dinterSize);
    matrixReduction<<<numY,numH,numH*sizeof(float)>>>(dinter,dy);
    sigmoidKernel<<<1,numY>>>(dy);

    //second layer done

    //backpropagation:
    

    backPropagationFirstKernel<<<numY,numH>>>(dh,dy,dyCorrect,dWeights[1],ddeltas[1],ddels,alpha,lrate);
    //dim3 grid(numY,numH);
    backPropagationSecondKernelPart1<<<numY,numH>>>(dh,dgammas,dWeights[1],ddels,alpha,lrate);
    matrixReduction<<<numH,numY,numY*sizeof(float)>>>(dgammas,dgammas);
    backPropagationSecondKernelPart2<<<numH,numX>>>(dx,dgammas,dWeights[0],ddeltas[0],alpha,lrate);






}