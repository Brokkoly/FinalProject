#ifdef _WIN32
#  define NOMINMAX 
#endif

#include "kernels.cu"
#include <stdio.h>
#include <stdlib.h>
int hibit(unsigned int n) {
    n |= (n >>  1);
    n |= (n >>  2);
    n |= (n >>  4);
    n |= (n >>  8);
    n |= (n >> 16);
    return n - (n >> 1);
}


unsigned char* read_array(const char* filename, int len) {
    
    FILE *fp = fopen(filename, "r");
    int temp;
    fscanf(fp,"%d",&temp);
    if(temp<len)len=temp;
    char *x = (char*) malloc(len * sizeof(char));
    for (int i = 0; i < len; i++) {
        fscanf(fp, "%f", &x[i]);
    }
    fclose(fp);
    return x;
}
unsigned char* read_array(const char* filename, int len,int rows,int cols) {
    FILE *fp = fopen(filename, "r");
    int temp;
    fscanf(fp,"%d",&temp);
    if(temp<len) len=temp;
    fscanf(fp,"%d",&rows);
    fscanf(fp,"%d",&cols);
    char *x = (char*) malloc(len * sizeof(char));
    for (int i = 0; i < len*rows*cols; i++) {
        fscanf(fp, "%f", &x[i]);
    }
    fclose(fp);
    return x;
}


int main(int argc,char** argv){




    //Initialize weight matrices

    //get inputs from training file
    //get inputs from test file

    //todo: add main
    float* a = (float*) malloc(2*13*sizeof(float));
    float* b = (float*) malloc(2);
    for(int i = 0; i < 13;i++){
        a[i] = i;
        b[0] +=i;
        a[i+13] = i;
        b[1]+=i;
    }
    a[13] +=100;
    b[1]+=100;
    float* da;
    printf("hibit: %x\n",hibit(13));
    hipMalloc(&da,sizeof(float)*26);
    hipMemcpy(da,a,sizeof(float)*26,hipMemcpyHostToDevice);
    matrixReductionDestructive<<<2,13,13*sizeof(float)>>>(da,13,hibit(13)<<1);
    hipMemcpy(a,da,sizeof(float)*26,hipMemcpyDeviceToHost);

    printf("Device Results: %f,%f\n",a[0],a[13]);
    printf("Host Results: %f,%f\n",b[0],b[1]);
    hipFree(da);
    free(a);
    free(b);
}


void trainingInstance(float* dx,float* dh, float* dy,float* dyCorrect,float* ddels,float* dgammas,float* dinter,float** dWeights,float** ddeltas,int numX,int numH,int numY,float alpha,float lrate,int dinterSize){

    //firstLayer
    forwardPropagation<<<numH,numX>>>(dx,dinter,dWeights[0],dinterSize);
    matrixReduction<<<numH,numX,numX*sizeof(float)>>>(dinter,dh,1024,hibit(1024));
    sigmoidKernel<<<1,numH>>>(dh);

    //first layer done

    //second layer:
    forwardPropagation<<<numY,numH>>>(dh,dinter,dWeights[1],dinterSize);
    matrixReduction<<<numY,numH,numH*sizeof(float)>>>(dinter,dy,1024,hibit(1024));
    sigmoidKernel<<<1,numY>>>(dy);

    //second layer done

    //backpropagation:
    

    backPropagationFirstKernel<<<numY,numH>>>(dh,dy,dyCorrect,dWeights[1],ddeltas[1],ddels,alpha,lrate);
    //dim3 grid(numY,numH);
    backPropagationSecondKernelPart1<<<numY,numH>>>(dh,dgammas,dWeights[1],ddels,alpha,lrate);
    matrixReduction<<<numH,numY,numY*sizeof(float)>>>(dgammas,dgammas,numY,hibit(numY));
    backPropagationSecondKernelPart2<<<numH,numX>>>(dx,dgammas,dWeights[0],ddeltas[0],alpha,lrate);






}
