#include "hip/hip_runtime.h"



#include <math.h>



//LSTM Kernels


//Initial: ft = sigmoid(W_f*concatenate(h_t-1,x_t]+bf))
__device__ float sigmoid(float x){
    float retVal = 0;
    retval = 1+exp(x);
    retval = 1/retval;
    return retval;
}
__device__ float sigmoidDerivative(float x){
    float val = sigmiod(x);
    return val*(1-val);
}
__global__ void forwardPropagationKernel(float* input,float* weights,float* output,int n, int m){
    int windex = threadIDx.x+blockIDx.x*blockDim.x;
    int vindex = threadIDx.x;
    output[windex] = input[vindex]+weights[windex];
}
__global__ void applySigmoid(float* hiddenLayer){
    int index = threadIDx.x;
    hiddenLayer[index] = sigmoid(hiddenLayer[index]);
}
__global__ void backPropagationFirstKernel(float* inputLayer,float* outputLayer,float* outputLayerTrue,float* weights,float* deltas,float* dels,int iSize,int oSize,float alpha,float lrate){
    int tindex = threadIDx.x;
    int oindex = blockIDx.x;
    int windex = tindex+blockIDx.x*blockDim.x;
    float del = (outputLayer[oindex]-outputLayerTrue[oindex])*outputLayer[oindex]*(1-outputLayer[oindex]);
    deltas[windex] = (1-alpha)*lrate*del*inputLayer[tindex]+alpha*deltas[windex];
    weights[windex] = weights[windex]-deltas[windex];
}

//blocks: (# of output layers,# of hidden layers)
__global__ void backPropagationSecondKernelPart1(float* hiddenLayer,float* gammas,float* weights,float* dels,int xSize,int hSize,int oSize,float alpha,float lrate){
    int tindex = threadIDx.x;//For each hidden node
    int oindex = blockIDx.x;//for each output node
    int mindex = blockIDx.y;//for each hidden layer
    int windex = tindex+blockIDx.x*blockDim.x;
    //int sindex = tindex+blockIDx.y*gridDim.x*blockDim.x;
    //float gamma = 0;
    gammas[gindex] = dels[oindex]*weights[windex]*h[mindex]*(1-h[mindex]);
    //float del = (outputLayer[oindex]-outputLayerTrue[oindex])*outputLayer[oindex]*(1-outputLayer[oindex]);
    //deltas[windex] = (1-alpha)*lrate*del*inputLayer[tindex]+alpha*deltas[windex];
    //weights[windex] = weights[windex]-deltas[windex];
}
__global__ void backPropagationSecondKernelPart1(float* gammas,int kSize){
    int tindex = threadIDx.x;//For each hidden node
    int gindex = tindex*blockDim.x;
    for(int i = 1; i < kSize;i++){
        gammas[gindex]+=gammas[gindex+i];
    }
}

__global__ void backPropagationSecondKernelPart2(float* inputLayer,float* gammas,float* weights,float* deltas,float alpha,float lrate){
    int tindex = threadIDx.x;//For each upper node
    int oindex = blockIDx.x;//for each hidden node
    int windex = tindex+blockIDx.x*blockDim.x;//
    deltas[windex] = (1-alpha)*lrate*gammas[oindex]*inputLayer[tindex]+alpha*deltas[windex];
    weights[windex] = weights[windex]-deltas[windex];
}
__global__ void forwardPropagation(float* x,float*Y,float* W,int yWidth){
    int tindex = threadIDx.x;
    int yindex = threadIDx.x+blockIDx.x*yWidth;
    int windex = threadIDx.x+blockIDx.x*blockDim.x;
    Y[yindex] = x[tindex]*W[windex];
}

__global__ void matrixReduction(float *g_data,float *out_data)
{
    extern __shared__ float sdata[];
    unsigned int tindex = (threadIdx.x);
    unsigned int i = blockIdx.x*blockDim.x+threadIdx.x;
    sdata[tindex] = g_data[i];
    __syncthreads();
    for(unsigned int s = blockDim.x/2;s>0;s>>=1){
        //int index = 2*s*tindex;
        if(tindex<s){
            sdata[tindex]+=sdata[tindex+s];
        }
        __syncthreads();
    }
    if(tindex==0) out_data[blockIdx.x] = sdata[0];
}

__global__ void firstStep(Matrix X, Matrix H,Matrix Inter,Matrix Wf,Matrix Wi,Matrix Wc,Matrix ft,Matrix it,Matrix Ct1,Matrix ot){

    int threadidx = threadIDx.x;// N threads, N values in output
    int blockId = blockIDx.x;
    int t = blockIDx.y;
    int function = blockIDx.y;
    int Tindex = t*blockDim.x*blockDim.y;
    int Iindex = blockId*blockDim.x*blockDim.y;
    int Windex = threadidx + blockIDx.x*blockDim.x+block
    if(function==0){
        //f_t
        for(int i=0; i < N;i++){

            Inter[Iindex]=Wf[i]
        }


    }
    //Things to calculate:
    //f_t
    //i_t
    //C_t
    //o_t



}