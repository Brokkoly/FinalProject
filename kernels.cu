#include "hip/hip_runtime.h"



#include <math.h>



//LSTM Kernels


//Initial: ft = sigmoid(W_f*concatenate(h_t-1,x_t]+bf))
__device__ float sigmoid(float x){
    float retVal = 0;
    retval = 1+exp(x);
    retval = 1/retval;
    return retval;
}

__global__ void firstStep(Matrix X, Matrix H,Matrix Inter,Matrix Wf,Matrix Wi,Matrix Wc,Matrix ft,Matrix it,Matrix Ct1,Matrix ot){

    int threadidx = threadIDx.x;// N threads, N values in output
    int blockId = blockIDx.x;
    int t = blockIDx.y;
    int function = blockIDx.y;
    int Tindex = t*blockDim.x*blockDim.y;
    int Iindex = blockId*blockDim.x*blockDim.y;
    int Windex = threadidx + blockIDx.x*blockDim.x+block
    if(function==0){
        //f_t
        for(int i=0; i < N;i++){

            Inter[Iindex]=Wf[i]
        }


    }
    //Things to calculate:
    //f_t
    //i_t
    //C_t
    //o_t



}